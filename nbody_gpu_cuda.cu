#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "timer.h"

#define SOFTENING 1e-9f
#define MAX_BLOCKS_PER_DIM 65535
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

typedef struct { float x, y, z, vx, vy, vz; } Body;

void randomizeBodies(Body *p, int n) {
    for (int i = 0; i < n; i++) {
        p[i].x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        p[i].y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        p[i].z = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        p[i].vx = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        p[i].vy = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        p[i].vz = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__ void bodyForce(Body *p, float dt, int n) {
  int tid0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = tid0; i < n; i += blockDim.x * gridDim.x) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = 1.0f / sqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }
    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
  for (int i = tid0 ; i < n; i += blockDim.x * gridDim.x) { // integrate position
    p[i].x = p[i].x + p[i].vx*dt;
    p[i].y = p[i].y + p[i].vy*dt;
    p[i].z = p[i].z + p[i].vz*dt;
  }
}

int particle_positions_to_csv(FILE *datafile, int iter, Body *p, int nBodies) {
    for (int i = 0 ; i < nBodies; i++) {
        fprintf(datafile, "%i, %f, %f, %f\n", iter, p[i].x, p[i].y, p[i].z);
    }
    return 0;
}

void calcPosition(Body *p, float dt, int nBodies, int nIters, int nthreads_per_block)
{
  FILE *datafile = fopen("nbody.csv","w");
  double totalTime = 0.0;
  Body *p_cu;
  hipMalloc((void**)&p_cu, nBodies*sizeof(Body));
  hipMemset(p_cu, 0.0, nBodies*sizeof(Body));
  hipMemcpy(p_cu, p, nBodies*sizeof(Body), hipMemcpyHostToDevice);
  StartTimer();
  for (int iter = 1; iter <= nIters; iter++) {
    int nblocks = MIN(nBodies / nthreads_per_block + 1, MAX_BLOCKS_PER_DIM);


    bodyForce<<<nblocks,nthreads_per_block>>>(p_cu, dt, nBodies);       // compute interbody forces
    
    const double tElapsed = GetTimer() / 1000.0;
    if (iter > 1) {                      // First iter is warm up
      totalTime += tElapsed; 
    }

    // hipMemcpy(p, p_cu, nBodies*sizeof(Body), hipMemcpyDeviceToHost);
    // if(iter%100 == 0){
    // particle_positions_to_csv(datafile, iter, p, nBodies);
    // }
  }
  // totalTime = totalTime/1000;
  double avgTime = totalTime / (double)(nIters-1);

  printf("avgTime: %.10f   totTime: %.10f \n", avgTime, totalTime);
  fclose(datafile);
}

int main(const int argc, const char** argv) {
  int nBodies;
  int nthreads_per_block;
  if (argc > 1) nBodies = atoi(argv[1]);
  if (argc > 2) nthreads_per_block = atoi(argv[2]);

  float dt = 0.01f; // time step
  int nIters = 20;  // simulation iterations

  int bytes = nBodies*sizeof(Body);
  float *buf = (float*)malloc(bytes);
  Body *p = (Body*)buf;

  randomizeBodies(p, nBodies); // Init pos / vel data
  calcPosition(p, dt, nBodies, nIters, nthreads_per_block);

  free(buf);
}
